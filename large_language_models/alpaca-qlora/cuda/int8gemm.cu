#include <iostream>
#include <torch/types.h>
#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/half.h>

#include <cutlass/gemm/device/gemm.h>
#include <cutlass/numeric_types.h>
#include <cutlass/util/host_tensor.h>

using namespace std;

// used by out_proj and fc2, return FP32
void int8gemm_cuda(
    torch::Tensor input, torch::Tensor weight, torch::Tensor out, //torch::Tensor bias,
    float alpha, float beta)
{
  // Define M, N, K
  auto M = input.size(0);
  auto N = weight.size(0); // oc, ic
  auto K = input.size(1);

  // Define dtype
  using ElementOutput = float;
  using ElementAccumulator = int32_t;
  using ElementComputeEpilogue = float;
  using ElementInputA = int8_t; // <- data type of elements in input matrix A
  using ElementInputB = int8_t; // <- data type of elements in input matrix B

  // The code section below describes matrix layout of input and output
  // matrices. Column Major for Matrix A, Row Major for Matrix B and Row Major
  // for Matrix C
  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

#if CUDA_ARCH >= 750
  using DefaultGemmCfg = cutlass::gemm::device::DefaultGemmConfiguration<
      cutlass::arch::OpClassTensorOp,
      cutlass::arch::Sm75,
      ElementInputA,
      ElementInputB,
      ElementOutput,
      ElementAccumulator>;
  using Gemm = cutlass::gemm::device::Gemm<
      ElementInputA,
      LayoutInputA,
      ElementInputB,
      LayoutInputB,
      ElementOutput,
      LayoutOutput,
      ElementAccumulator,
      cutlass::arch::OpClassTensorOp,
      cutlass::arch::Sm75,
      DefaultGemmCfg::ThreadblockShape,
      DefaultGemmCfg::WarpShape,
      DefaultGemmCfg::InstructionShape,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput,
          128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator,
          ElementComputeEpilogue>>;
#else
  #error "Unsupported cuda arch"
#endif

  auto input_size = cutlass::MatrixCoord(M, K);
  auto weight_size = cutlass::MatrixCoord(K, N);
  auto output_size = cutlass::MatrixCoord(M, N);

  // constexpr int kSparse = Gemm::kSparse;
  // How many elements of A are covered per ElementE
  // constexpr int kElementsPerElementE = Gemm::kElementsPerElementE;
  // The size of individual meta data
  // constexpr int kMetaSizeInBits = Gemm::kMetaSizeInBits;
  cutlass::gemm::GemmCoord problem_size(M, N, K);

  cutlass::TensorRef<ElementInputA, LayoutInputA> input_ref(
      input.data_ptr<ElementInputA>(), LayoutInputA::packed(input_size));
  cutlass::TensorRef<ElementInputB, LayoutInputB> weight_ref(
      weight.data_ptr<ElementInputB>(), LayoutInputB::packed(weight_size));
  cutlass::TensorRef<ElementOutput, LayoutOutput> out_ref(
      out.data_ptr<ElementOutput>(), LayoutOutput::packed(output_size));

  typename Gemm::Arguments arguments{
      problem_size, // <- problem size of matrix multiplication
      input_ref,    // <- reference to matrix A on device
      weight_ref,   // <- reference to matrix B on device
      out_ref,      // <- reference to matrix C on device
      out_ref,      // <- reference to matrix D on device
      {alpha, beta}, 1};
  Gemm gemm_op;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm_op.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm_op(); // execute
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot run");
  }

  return; //out;
}
