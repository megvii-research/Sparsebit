#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using at::cuda::getCurrentCUDAStream;

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const scalar_t *__restrict__ inp1,
    const int *__restrict__ inp2,
    scalar_t *__restrict__ out,
    const scalar_t *__restrict__ scales,
    const scalar_t *__restrict__ zeros,
    int height,
    int width,
    int inchannels,
    int batch,
    int group_size);

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVUP(a, b) (((a) + (b)-1) / (b))
#define BIT 4
#define BLOCKWIDTH_4BIT 64
#define BLOCKHEIGHT_4BIT 16
#define BLOCKLEN_4BIT (BLOCKHEIGHT_4BIT * 32 / BIT)
#define READ_REPEAT_NUM (BLOCKLEN_4BIT / BLOCKWIDTH_4BIT)

#define BLOCKWIDTH BLOCKWIDTH_4BIT
#define BLOCKHEIGHT BLOCKHEIGHT_4BIT
#define BLOCKLEN BLOCKLEN_4BIT

#define DIM_LEN(x) ((x).sizes().vec().size())

void vecquant4matmul_cuda(
    torch::Tensor inp1,
    torch::Tensor inp2,
    torch::Tensor out,
    torch::Tensor scales,
    torch::Tensor zeros,
    int group_size = 0)
{
    TORCH_CHECK(DIM_LEN(inp1) >= 2, "input1 must be with dimension >= 2");
    int inchannels = inp1.size(-1);
    int batch = inp1.numel() / inchannels;

    TORCH_CHECK(DIM_LEN(inp2) == 2, "input2 must be with dimension == 2");
    int height = inp2.size(0);
    int width = inp2.size(1);

    TORCH_CHECK(out.size(-1) == width, "output channel must be the same with input2 out_channel");
    dim3 blocks(
        (height + BLOCKHEIGHT - 1) / BLOCKHEIGHT,
        (width + BLOCKWIDTH - 1) / BLOCKWIDTH);
    dim3 threads(BLOCKWIDTH);

    if (group_size != 0)
    {
        TORCH_CHECK(group_size / 128 * 128 == group_size, "only group_size divisible by 128 is supported in 4-bit quantization");
    }
    else
        group_size = inchannels;

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES(
        inp1.type(), "vecquant4matmul_cuda",
        ([&]
         { VecQuant4MatMulKernel<<<blocks, threads, BLOCKHEIGHT * BLOCKWIDTH * sizeof(unsigned) + BLOCKLEN * 2 * sizeof(scalar_t), stream>>>(
               inp1.data<scalar_t>(),
               inp2.data<int>(),
               out.data<scalar_t>(),
               scales.data<scalar_t>(),
               zeros.data<scalar_t>(),
               height,
               width,
               inchannels,
               batch,
               group_size); }));
}

__device__ inline unsigned int as_unsigned(int i)
{
    return *reinterpret_cast<unsigned int *>(&i);
}

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const scalar_t *__restrict__ inp1,
    const int *__restrict__ inp2,
    scalar_t *__restrict__ out,
    const scalar_t *__restrict__ scales,
    const scalar_t *__restrict__ zeros,
    int height,
    int width,
    int inchannels,
    int batch,
    int group_size)
{
    __shared__ unsigned int weight[BLOCKHEIGHT * BLOCKWIDTH];
    __shared__ scalar_t inputs[BLOCKLEN << 1];

    int row = BLOCKHEIGHT * blockIdx.x;
    int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;
    int inp1_pos = BLOCKLEN * blockIdx.x + threadIdx.x;

    int cur = 0, pre = BLOCKLEN;
    auto cur_input = inputs + cur, pre_input = inputs + pre;
    int group_max = DIVUP(inchannels, group_size);
    int group_pos = row * 32 / 4 / group_size;
    scalar_t scale = col < width ? scales[group_max * col + group_pos] : (scalar_t)0;
    scalar_t zero = col < width ? zeros[group_max * col + group_pos] : (scalar_t)0;
    scalar_t res;

    int block_height = col < width ? MIN(BLOCKHEIGHT, height - row) : 0;
    int block_height8 = block_height * 8;
    int read_times = MIN(READ_REPEAT_NUM, (inchannels - inp1_pos + BLOCKWIDTH - 1) / BLOCKWIDTH);

    for (int i = width * row + col, j = 0; j < block_height; j += 1, i += width)
        weight[j * BLOCKWIDTH + threadIdx.x] = as_unsigned(inp2[i]);

    for (int i = 0; i < READ_REPEAT_NUM; ++i)
        if (i < read_times)
            pre_input[i * BLOCKWIDTH + threadIdx.x] = inp1[inp1_pos + i * BLOCKWIDTH];
        else
        {
            pre_input[i * BLOCKWIDTH + threadIdx.x] = 0;
            cur_input[i * BLOCKWIDTH + threadIdx.x] = 0;
        }

    __syncthreads();
    int output_pos = col;
    for (int b = 1; b < batch; ++b)
    {
        cur = BLOCKLEN - cur;
        pre = BLOCKLEN - pre;
        cur_input = inputs + cur;
        pre_input = inputs + pre;
        // extract b-th data
        for (int i = 0; i < read_times; ++i)
            pre_input[i * BLOCKWIDTH + threadIdx.x] = inp1[b * inchannels + i * BLOCKWIDTH + inp1_pos];

        // (b - 1) - th calculation
        res = 0;
        for (int k = 0, i = threadIdx.x; k < block_height8; k += 8, i += BLOCKWIDTH)
        {
            res += (scale * scalar_t((weight[i] >> 0) & 0xF) - zero) * cur_input[k | 0];
            res += (scale * scalar_t((weight[i] >> 4) & 0xF) - zero) * cur_input[k | 1];
            res += (scale * scalar_t((weight[i] >> 8) & 0xF) - zero) * cur_input[k | 2];
            res += (scale * scalar_t((weight[i] >> 12) & 0xF) - zero) * cur_input[k | 3];
            res += (scale * scalar_t((weight[i] >> 16) & 0xF) - zero) * cur_input[k | 4];
            res += (scale * scalar_t((weight[i] >> 20) & 0xF) - zero) * cur_input[k | 5];
            res += (scale * scalar_t((weight[i] >> 24) & 0xF) - zero) * cur_input[k | 6];
            res += (scale * scalar_t((weight[i] >> 28) & 0xF) - zero) * cur_input[k | 7];
        }
        if (col < width)
            atomicAdd(out + output_pos, res);
        output_pos += width;
        __syncthreads();
    }
    // do batch - th calculation

    cur_input = inputs + pre;
    res = 0;
    for (int k = 0, i = threadIdx.x; k < block_height8; k += 8, i += BLOCKWIDTH)
    {
        res += (scale * scalar_t((weight[i] >> 0) & 0xF) - zero) * cur_input[k | 0];
        res += (scale * scalar_t((weight[i] >> 4) & 0xF) - zero) * cur_input[k | 1];
        res += (scale * scalar_t((weight[i] >> 8) & 0xF) - zero) * cur_input[k | 2];
        res += (scale * scalar_t((weight[i] >> 12) & 0xF) - zero) * cur_input[k | 3];
        res += (scale * scalar_t((weight[i] >> 16) & 0xF) - zero) * cur_input[k | 4];
        res += (scale * scalar_t((weight[i] >> 20) & 0xF) - zero) * cur_input[k | 5];
        res += (scale * scalar_t((weight[i] >> 24) & 0xF) - zero) * cur_input[k | 6];
        res += (scale * scalar_t((weight[i] >> 28) & 0xF) - zero) * cur_input[k | 7];
    }

    if (col < width)
        atomicAdd(out + output_pos, res);
}

#undef BLOCKWIDTH
#undef BLOCKHEIGHT
